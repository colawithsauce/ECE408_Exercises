#include "hip/hip_runtime.h"
#include "../cuda_alias.h"
#include <bits/types/clock_t.h>
#include <chrono>
#include <cstdio>
#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime.h>
#include <>
#include <locale>

__global__ void
matrix_multi_kernel(double* A_d, double* B_d, double* C_d, int M, int N, int S)
{
    // A_d is M x S, while B_d is S x N
    unsigned int col = threadIdx.x + blockDim.x * blockIdx.x;
    unsigned int row = threadIdx.y + blockDim.y * blockIdx.y;

    if (row < M && col < N) {
        double sum = 0;
        for (int i = 0; i != S; i++) {
            sum += A_d[row * S + i] * B_d[i * S + col];
        }

        C_d[row * N + col] = sum;
    }
}

extern __global__ void
matrix_multi_tile_kernel(const double* A_d, const double* B_d, double* C_d, int M, int N, int S, int nSZa, int nBlkWidth);

hipError_t
matrix_multi(const double* A_h, const double* B_h, double* C_h_2, int M, int N, int S)
{
    double *A_d, *B_d, *C_d, *C_d1;
    clock_t start = clock();
    double elapsed = 0;
    hipError_t err = hipSuccess;

    dim3 dimGrid = { (unsigned int)ceil(N / 128.0), (unsigned int)ceil(M / 128.0),
        1 };
    dim3 dimBlock = { 128, 128, 1 };

    err = hipMalloc((void**)&A_d, M * S * sizeof(double));
    CUDA_CHECK(err, "Can't hipMalloc");

    err = hipMalloc((void**)&B_d, N * S * sizeof(double));
    CUDA_CHECK(err, "Can't hipMalloc");

    err = hipMalloc((void**)&C_d, M * N * sizeof(double));
    CUDA_CHECK(err, "Can't hipMalloc");

    err = hipMalloc((void**)&C_d1, M * N * sizeof(double));
    CUDA_CHECK(err, "Can't hipMalloc");

    err = hipMemcpy(A_d, A_h, M * S * sizeof(double), hipMemcpyHostToDevice);
    CUDA_CHECK(err, "Can't hipMemcpy");
    err = hipMemcpy(B_d, B_h, N * S * sizeof(double), hipMemcpyHostToDevice);
    CUDA_CHECK(err, "Can't hipMemcpy");

    // compare the time consume
    // first
    start = clock();
    matrix_multi_kernel KERNEL_ARGS2(dimGrid, dimBlock)(A_d, B_d, C_d, M, N, S);
    elapsed = 1000 * (double)(clock() - start) / CLOCKS_PER_SEC; // in milliseconds
    printf("normal matrix_multi: %lf ms\n", elapsed);

    err = hipMemcpy(C_h_2, C_d, M * N * sizeof(double), hipMemcpyDeviceToHost);
    CUDA_CHECK(err, "Can't hipMemcpy");

    // second
    start = clock();
    matrix_multi_tile_kernel KERNEL_ARGS3(dimGrid, dimBlock, dimBlock.x * 2)(A_d, B_d, C_d, M, N, S, dimBlock.x, dimBlock.x);
    elapsed = 1000 * (double)(clock() - start) / CLOCKS_PER_SEC; // in milliseconds
    printf("normal matrix_multi: %lf ms\n", elapsed);

    err = hipMemcpy(C_h_2 + M * N, C_d, M * N * sizeof(double), hipMemcpyDeviceToHost);
    CUDA_CHECK(err, "Can't hipMemcpy");
Error:
    hipFree(A_d);
    hipFree(B_d);
    hipFree(C_d);

    return err;
}
