#include "hip/hip_runtime.h"
#include "../cuda_alias.h"
#include <bits/types/clock_t.h>
#include <cassert>
#include <chrono>
#include <cstdio>
#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>

#define TILE_WIDTH 2

// @param:
//   A_d: M * S matrix
//   B_d: S * N matrix
//   C_d: M * N matrix
__global__ void
matrix_multi_tile_kernel(const double* A_d, const double* B_d, double* C_d, int M, int N, int S, int nSZa, int nBlkWidth)
{
    // Initialize space for tiling multiplication
    extern __shared__ double Ads_Bds[]; // Defined in the kernel arguments
    double* Ads = Ads_Bds;
    double* Bds = Ads_Bds + nSZa;

    const int bx = blockIdx.x;
    const int by = blockIdx.y;
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;

    int Row = nBlkWidth * by + threadIdx.y;
    int Col = nBlkWidth * bx + threadIdx.x;

    // do matrix multiplication
    double Cvalue = 0;
    for (int ph = 0; ph < ceil(max((float)M / nBlkWidth, (float)N / nBlkWidth)); ph++) {
        if (Row < M && (ph * nBlkWidth + tx) < S) {
            Ads[ty * nBlkWidth + tx] = A_d[Row * S + (ph * nBlkWidth + tx)];
        } else {
            Ads[ty * nBlkWidth + tx] = 0;
        }

        if (Col < N && (ph * nBlkWidth + ty) < S) {
            Bds[ty * nBlkWidth + tx] = B_d[(ph * nBlkWidth + ty) * N + Col];
        } else {
            Bds[ty * nBlkWidth + tx] = 0;
        }

        __syncthreads();

        // Accumulate the result in this phase
        for (int i = 0; i != nBlkWidth; i++) {
            Cvalue += Ads[ty * nBlkWidth + i] * Bds[i * nBlkWidth + tx];
        }

        __syncthreads();
    }

    C_d[(Row * N) + Col] = Cvalue;
}

__global__ void
matrix_multi_tile_simple_kernel(const double* A_d, const double* B_d, double* C_d, int width)
{
    __shared__ double Ads[TILE_WIDTH][TILE_WIDTH];
    __shared__ double Bds[TILE_WIDTH][TILE_WIDTH];

    const int bx = blockIdx.x;
    const int by = blockIdx.y;
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;

    int Row = TILE_WIDTH * by + threadIdx.y;
    int Col = TILE_WIDTH * bx + threadIdx.x;

    double Cvalue = 0.0;
    for (int ph = 0; ph < width / (float)TILE_WIDTH; ph++) {
        Ads[ty][tx] = A_d[Row * width + ph * TILE_WIDTH + tx];
        Bds[ty][tx] = B_d[(ph * TILE_WIDTH + ty) * width + Col];

        __syncthreads();

        for (int k = 0; k < TILE_WIDTH; k++) {
            Cvalue += Ads[ty][k] * Bds[k][tx];
        }

        __syncthreads();
    }

    C_d[Row * width + Col] = Cvalue;
}

hipError_t
matrix_multi_tile(const double* A_h, const double* B_h, double* C_h, int M, int N, int S)
{
    double *A_d, *B_d, *C_d;
    hipError_t err = hipSuccess;

    // dim3 dimGrid = { (unsigned int)ceil(N / 32.0), (unsigned int)ceil(M / 32.0), 1 };
    // dim3 dimBlock = { 32, 32, 1 };

    dim3 dimGrid = { (unsigned int)ceil(N / 2.0), (unsigned int)ceil(M / 2.0), 1 };
    dim3 dimBlock = { 2, 2, 1 };

    err = hipMalloc((void**)&A_d, M * S * sizeof(double));
    CUDA_CHECK(err, "Can't hipMalloc");

    err = hipMalloc((void**)&B_d, N * S * sizeof(double));
    CUDA_CHECK(err, "Can't hipMalloc");

    err = hipMalloc((void**)&C_d, M * N * sizeof(double));
    CUDA_CHECK(err, "Can't hipMalloc");

    err = hipMemcpy(A_d, A_h, M * S * sizeof(double), hipMemcpyHostToDevice);
    CUDA_CHECK(err, "Can't hipMemcpy");
    err = hipMemcpy(B_d, B_h, N * S * sizeof(double), hipMemcpyHostToDevice);
    CUDA_CHECK(err, "Can't hipMemcpy");

    // matrix_multi_tile_kernel<<<dimGrid, dimBlock, 1024 * 2 * sizeof(double)>>>(A_d,
    //     B_d,
    //     C_d,
    //     M,
    //     N,
    //     S,
    //     1024,
    //     32);
    matrix_multi_tile_simple_kernel<<<dimGrid, dimBlock>>>(A_d, B_d, C_d, 2);

    err = hipGetLastError();
    CUDA_CHECK(err, "Can't launch kernel matrix_multi_tile_kernel");

    err = hipMemcpy(C_h, C_d, M * N * sizeof(double), hipMemcpyDeviceToHost);
    CUDA_CHECK(err, "Can't hipMemcpy");

Error:
    hipFree(A_d);
    hipFree(B_d);
    hipFree(C_d);

    return err;
}
