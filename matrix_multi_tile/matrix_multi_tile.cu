#include <cmath>
#include <hip/hip_runtime.h>


#define TILE_WIDTH 16

// @param:
//   A_d: M * S matrix
//   B_d: S * N matrix
//   C_d: M * N matrix
//   And we assume that blockDim.x == blocDim.y && blockDim.x == TILE_WIDTH
__global__ void
matrix_multi_tile_kernel(const double* A_d, const double* B_d, double* C_d, int M, int N, int S, int nSZa, int nBlkWidth)
{
    // Initialize space for tiling multiplication
    extern __shared__ double Ads_Bds[]; // Defined in the kernel arguments
    double* Ads = Ads_Bds;
    double* Bds = Ads_Bds + nSZa;

    const int bx = blockIdx.x;
    const int by = blockIdx.y;
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;

    int Row = nBlkWidth * by + threadIdx.y;
    int Col = nBlkWidth * bx + threadIdx.x;

    // do matrix multiplication
    double Cvalue = 0;
    for (int ph = 0; ph < ceil(max((float)M / nBlkWidth, (float)N / nBlkWidth)); ph++) {
        if (Row < M && (ph * nBlkWidth + tx) < S) {
            Ads[ty * nBlkWidth + tx] = A_d[Row * S + (ph * nBlkWidth + tx)];
        } else {
            Ads[ty * nBlkWidth + tx] = 0;
        }

        if (Col < N && (ph * nBlkWidth + ty) < S) {
            Bds[ty * nBlkWidth + tx] = B_d[(ph * nBlkWidth + ty) * N + Col];
        } else {
            Bds[ty * nBlkWidth + tx] = 0;
        }

        __syncthreads();

        // Accumulate the result in this phase
        for (int i = 0; i != nBlkWidth; i++) {
            Cvalue += Ads[ty * nBlkWidth + i] * Bds[i * nBlkWidth + tx];
        }

        __syncthreads();
    }

    C_d[(Row * N) + Col] = Cvalue;
}
