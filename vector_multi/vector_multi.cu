#include "hip/hip_runtime.h"
#include "../cuda_alias.h"
#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime.h>
#include <>
#include <hip/driver_types.h>
#include <stdio.h>
#include <time.h>

// Caculate A = B * v, where B is M x N, v is N x 1, and A is M x 1
__global__ void vector_multi(float *A_d, float *B_d, float *v_d, int M, int N)
{
    unsigned int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < M)
    {
        float sum = 0;
        for (unsigned int x = 0; x != N; ++x)
        {
            sum += B_d[i * N + x] * v_d[x];
        }

        A_d[i] = sum;
    }
}

// launcher of vector_multi
hipError_t vector_multi_launcher(float *A_h, const float *B_h, const float *v_h, int M, int N)
{
    float *A_d, *B_d, *v_d;
    hipError_t err = hipSuccess;
    clock_t start;
    float duration;

    err = hipMalloc((void **)&A_d, M * sizeof(float));
    CUDA_CHECK(err, "failed to malloc");

    err = hipMalloc((void **)&B_d, M * N * sizeof(float));
    CUDA_CHECK(err, "failed to malloc");

    err = hipMalloc((void **)&v_d, N * sizeof(float));
    CUDA_CHECK(err, "failed to malloc");

    err = hipMemcpy(B_d, B_h, M * N * sizeof(float), hipMemcpyHostToDevice);
    CUDA_CHECK(err, "failed to memcpy");

    err = hipMemcpy(v_d, v_h, N * sizeof(float), hipMemcpyHostToDevice);
    CUDA_CHECK(err, "failed to memcpy");

    start = clock();
    vector_multi<<<ceil(M / 32.0), 32>>>(A_d, B_d, v_d, M, N);
    duration = 1000.0 * (clock() - start) / CLOCKS_PER_SEC;
    printf("vector_multi: %f ms\n", duration);

    err = hipMemcpy(A_h, A_d, M * sizeof(float), hipMemcpyDeviceToHost);
    CUDA_CHECK(err, "failed to memcpy");

Error:
    hipFree(A_d);
    hipFree(B_d);
    hipFree(v_d);

    return err;
}
