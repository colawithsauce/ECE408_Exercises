#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <device_types.h>

#include "assert.h"
#include "cuda_alias.hpp"

#define BLOCK_DIM 64

/* NOTE: This is not an multi-block kernel. which means it can't handle more
 * than one block in put data.
 *
 * The multi-block version would be introduced later in this Chapter!!!
 *
 * NOTE: This kernel assume the size of input become the result of pow(2, n) and
 * that the blockDim.x become ~size / 2~.
 * */
__global__ void
reduction_kernel_1(float* in, float* out)
{
    unsigned int i = 2 * threadIdx.x;

    for (unsigned int stride = 1; stride <= blockDim.x; stride *= 2) {
        if (i % stride == 0) {
            in[i] += in[stride + i];
        }

        __syncthreads();
    }

    if (threadIdx.x == 0) {
        *out = in[0];
    }
}

/* NOTE: This is the enhanced version of kernel_1. It makes stride decrease by
 * each iteration from dimBlock to 1. Which makes great difference on control
 * divergence and memory coalescing.
 *
 * */
__global__ void
reduction_kernel_2(float* input, float* output)
{
    unsigned int i = threadIdx.x;
    for (unsigned int stride = blockDim.x; stride >= 1; stride /= 2) {
        if (i < stride) {
            input[i] += input[i + stride];
            __syncthreads();
        }
    }

    if (i == 0) {
        *output = input[0];
    }
}

/* NOTE: Enhance the former kernel with shared memory. */
__global__ void
reduction_kernel_3(float* input, float* output)
{
    __shared__ float input_s[BLOCK_DIM];
    assert(blockDim.x == BLOCK_DIM);

    unsigned int i = threadIdx.x;
    input_s[i] = input[i] + input[i + blockDim.x];
    for (unsigned int stride = blockDim.x / 2; stride >= 1; stride /= 2) {
        __syncthreads();
        if (i < stride) {
            input_s[i] += input_s[i + stride];
        }
    }

    if (i == 0) {
        *output = input_s[0];
    }
}

hipError_t
reduction_kernel_1_launcher(float* in, float* out, int size)
{
    dim3 dimBlock, dimGrid;
    hipError_t err = hipSuccess;

    float *in_d = nullptr, *out_d = nullptr;
    err = hipMalloc((void**)&in_d, sizeof(float) * size);
    CUDA_CHECK(err, "Malloc failed!");

    err = hipMalloc((void**)&out_d, sizeof(float) * 1);
    CUDA_CHECK(err, "Malloc failed!");

    err = hipMemcpy(
      (void*)in_d, (void*)in, sizeof(float) * size, hipMemcpyHostToDevice);
    CUDA_CHECK(err, "Memcpy failed!");

    err = hipMemcpy(
      (void*)out_d, (void*)out, sizeof(float) * 1, hipMemcpyHostToDevice);
    CUDA_CHECK(err, "Memcpy failed!");

    dimBlock = { (uint)size / 2, 1, 1 };
    dimGrid = { 1, 1, 1 };

    printf("Launching kernel with dimGrid %u ...\n", dimGrid.x);
    reduction_kernel_3<<<dimGrid, dimBlock>>>(in_d, out_d);

    err = hipGetLastError();
    CUDA_CHECK(err, "Error when calling kernel");

    hipMemcpy(out, out_d, sizeof(float), hipMemcpyDeviceToHost);

Error:
    hipFree(in_d);
    hipFree(out_d);
    return err;
}
