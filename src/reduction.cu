#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <device_types.h>

#include "cuda_alias.hpp"

/* NOTE: This is not an multi-block kernel. which means it can't handle more
 * than one block in put data.
 *
 * The multi-block version would be introduced later in this Chapter!!!
 *
 * NOTE: This kernel assume the size of input become the result of pow(2, n) and
 * that the blockDim.x become ~size / 2~.
 * */
__global__ void
reduction_kernel_1(float* in, float* out, int size)
{
    unsigned int i = 2 * threadIdx.x;

    for (unsigned int stride = 1; stride <= blockDim.x; stride *= 2) {
        if (i % stride == 0) {
            in[i] += in[stride + i];
        }

        __syncthreads();
    }

    if (threadIdx.x == 0) {
        *out = in[0];
    }
}

// __global__ void
// reduction_kernel_2(float* input, float* output)
// {
//     unsigned int i = threadIdx.x;
// }

hipError_t
reduction_kernel_1_launcher(float* in, float* out, int size)
{
    dim3 dimBlock, dimGrid;
    hipError_t err = hipSuccess;

    float *in_d = nullptr, *out_d = nullptr;
    err = hipMalloc((void**)&in_d, sizeof(float) * size);
    CUDA_CHECK(err, "Malloc failed!");

    err = hipMalloc((void**)&out_d, sizeof(float) * 1);
    CUDA_CHECK(err, "Malloc failed!");

    err = hipMemcpy(
      (void*)in_d, (void*)in, sizeof(float) * size, hipMemcpyHostToDevice);
    CUDA_CHECK(err, "Memcpy failed!");

    err = hipMemcpy(
      (void*)out_d, (void*)out, sizeof(float) * 1, hipMemcpyHostToDevice);
    CUDA_CHECK(err, "Memcpy failed!");

    dimBlock = { (uint)size / 2, 1, 1 };
    dimGrid = { 1, 1, 1 };

    printf("Launching kernel with dimGrid %u ...\n", dimGrid.x);
    reduction_kernel_1<<<dimGrid, dimBlock>>>(in_d, out_d, size);

    err = hipGetLastError();
    CUDA_CHECK(err, "Error when calling kernel");

    hipMemcpy(out, out_d, sizeof(float), hipMemcpyDeviceToHost);

Error:
    hipFree(in_d);
    hipFree(out_d);
    return err;
}
