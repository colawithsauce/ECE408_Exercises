#include "hip/hip_runtime.h"
#include <cassert>
#include <cstdio>
#include <ctime>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>

#include "cuda_alias.hpp"

__global__ void
matrix_multi_kernel(double* A_d, double* B_d, double* C_d, int M, int N, int S)
{
    // A_d is M x S, while B_d is S x N
    unsigned int col = threadIdx.x + blockDim.x * blockIdx.x;
    unsigned int row = threadIdx.y + blockDim.y * blockIdx.y;

    if (row < M && col < N) {
        double sum = 0;
        for (int i = 0; i != S; i++) {
            sum += A_d[row * S + i] * B_d[i * S + col];
        }

        C_d[row * N + col] = sum;
    }
}

hipError_t
matrix_multi(const double* A_h,
             const double* B_h,
             double* C_h,
             int M,
             int N,
             int S)
{
    double *A_d, *B_d, *C_d;
    hipError_t err = hipSuccess;

    int count = 0;
    hipGetDeviceCount(&count);
    printf("You have %d devices\n", count);

    dim3 dimGrid = { (unsigned int)ceil(N / 32.0),
                     (unsigned int)ceil(M / 32.0),
                     1 };
    dim3 dimBlock = { 32, 32, 1 };

    err = hipMalloc((void**)&A_d, M * S * sizeof(double));
    CUDA_CHECK(err, "Can't hipMalloc");

    err = hipMalloc((void**)&B_d, N * S * sizeof(double));
    CUDA_CHECK(err, "Can't hipMalloc");

    err = hipMalloc((void**)&C_d, M * N * sizeof(double));
    CUDA_CHECK(err, "Can't hipMalloc");

    err = hipMemcpy(A_d, A_h, M * S * sizeof(double), hipMemcpyHostToDevice);
    CUDA_CHECK(err, "Can't hipMemcpy");

    err = hipMemcpy(B_d, B_h, N * S * sizeof(double), hipMemcpyHostToDevice);
    CUDA_CHECK(err, "Can't hipMemcpy");

    matrix_multi_kernel KERNEL_ARGS2(dimGrid, dimBlock)(A_d, B_d, C_d, M, N, S);
    err = hipGetLastError();
    CUDA_CHECK(err, "ERROR when call");

    err = hipMemcpy(C_h, C_d, M * N * sizeof(double), hipMemcpyDeviceToHost);
    CUDA_CHECK(err, "Can't hipMemcpy");

Error:
    hipFree(A_d);
    hipFree(B_d);
    hipFree(C_d);

    return err;
}
