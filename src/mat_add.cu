#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>

#include "cuda_alias.hpp"

#define msg ""

__global__ void mat_add_kernel(const double* A_d, const double* B_d,
                               double* C_d, int M, int N) {
  int x = threadIdx.x + blockIdx.x * blockDim.x;
  int y = threadIdx.y + blockIdx.y * blockDim.y;

  if (y < M && x < N) {
    C_d[y * N + x] = A_d[y * N + x] + B_d[y * N + x];
  }
}

hipError_t mat_add(const double* A_h, const double* B_h, double* C_h, int M,
                    int N) {
  hipError_t err = hipSuccess;
  double *A_d, *B_d, *C_d;

  dim3 dimGrid = {(unsigned int)ceil(N / 32.0), (unsigned int)ceil(M / 32.0),
                  1};
  dim3 dimBlock = {32, 32, 1};

  err = hipMalloc((void**)&A_d, sizeof(double) * M * N);
  CUDA_CHECK(err, msg);

  err = hipMalloc((void**)&B_d, sizeof(double) * M * N);
  CUDA_CHECK(err, msg);

  err = hipMalloc((void**)&C_d, sizeof(double) * M * N);
  CUDA_CHECK(err, msg);

  err = hipMemcpy(A_d, A_h, sizeof(double) * M * N, hipMemcpyHostToDevice);
  CUDA_CHECK(err, msg);
  err = hipMemcpy(B_d, B_h, sizeof(double) * M * N, hipMemcpyHostToDevice);
  CUDA_CHECK(err, msg);

  mat_add_kernel<<<dimGrid, dimBlock>>>(A_d, B_d, C_d, M, N);
  err = hipGetLastError();
  CUDA_CHECK(err, "Launch kernel failed");

  err = hipMemcpy(C_h, C_d, sizeof(double) * M * N, hipMemcpyDeviceToHost);
  CUDA_CHECK(err, msg);

Error:
  hipFree(A_d);
  hipFree(B_d);
  hipFree(C_d);

  return err;
}
