#include "hip/hip_runtime.h"
#include "cuda_alias.hpp"

#define TILE_WIDTH 16

__global__ void
matrix_multi_tile_simple_kernel(const double* A_d,
                                const double* B_d,
                                double* C_d,
                                int width)
{
    __shared__ double Ads[TILE_WIDTH][TILE_WIDTH];
    __shared__ double Bds[TILE_WIDTH][TILE_WIDTH];

    const int bx = blockIdx.x;
    const int by = blockIdx.y;
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;

    int Row = TILE_WIDTH * by + threadIdx.y;
    int Col = TILE_WIDTH * bx + threadIdx.x;

    double Cvalue = 0.0;
    for (int ph = 0; ph < width / (float)TILE_WIDTH; ph++) {
        Ads[ty][tx] = A_d[Row * width + ph * TILE_WIDTH + tx];
        Bds[ty][tx] = B_d[(ph * TILE_WIDTH + ty) * width + Col];

        __syncthreads();

        for (int k = 0; k < TILE_WIDTH; k++) {
            Cvalue += Ads[ty][k] * Bds[k][tx];
        }

        __syncthreads();
    }

    C_d[Row * width + Col] = Cvalue;
}

hipError_t
matrix_multi_tile_simple(const double* A_h,
                         const double* B_h,
                         double* C_h,
                         int width)
{
    double *A_d, *B_d, *C_d;
    hipError_t err = hipSuccess;

    dim3 dimGrid = { (unsigned int)ceil(width / (float)TILE_WIDTH),
                     (unsigned int)ceil(width / (float)TILE_WIDTH),
                     1 };
    dim3 dimBlock = { TILE_WIDTH, TILE_WIDTH, 1 };

    err = hipMalloc((void**)&A_d, width * width * sizeof(double));
    CUDA_CHECK(err, "Can't hipMalloc");

    err = hipMalloc((void**)&B_d, width * width * sizeof(double));
    CUDA_CHECK(err, "Can't hipMalloc");

    err = hipMalloc((void**)&C_d, width * width * sizeof(double));
    CUDA_CHECK(err, "Can't hipMalloc");

    err = hipMemcpy(
      A_d, A_h, width * width * sizeof(double), hipMemcpyHostToDevice);
    CUDA_CHECK(err, "Can't hipMemcpy");
    err = hipMemcpy(
      B_d, B_h, width * width * sizeof(double), hipMemcpyHostToDevice);
    CUDA_CHECK(err, "Can't hipMemcpy");

    matrix_multi_tile_simple_kernel<<<dimGrid, dimBlock>>>(
      A_d, B_d, C_d, width);

    err = hipGetLastError();
    CUDA_CHECK(err, "Can't launch kernel matrix_multi_tile_kernel");

    err = hipMemcpy(
      C_h, C_d, width * width * sizeof(double), hipMemcpyDeviceToHost);
    CUDA_CHECK(err, "Can't hipMemcpy");

Error:
    hipFree(A_d);
    hipFree(B_d);
    hipFree(C_d);

    return err;
}
