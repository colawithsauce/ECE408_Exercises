#include "hip/hip_runtime.h"
#include <algorithm>
#include <assert.h>
#include <hip/hip_runtime_api.h>
#include <device_types.h>

#include "cuda_alias.hpp"

const int BLOCK_WIDTH = 16;

__global__ void
matTransposeKernel(double* in_d, double* out_d, int width, int height)
{
    int i = threadIdx.y + blockIdx.y * blockDim.y;
    int j = threadIdx.x + blockIdx.x * blockDim.x;

    if (j < width && i < height) {
        out_d[i + j * height] = in_d[i * width + j];
    }
}

hipError_t
matTranspose(double* in_h, double* out_h, int width, int height)
{
    double *in_d, *out_d;
    hipError_t err = hipSuccess;

    int T = std::max(width, height);

    dim3 dimBlock, dimGrid;
    dimBlock = { BLOCK_WIDTH, BLOCK_WIDTH, 1 };
    dimGrid = { (unsigned int)ceil((float)T / dimBlock.x),
                (unsigned int)ceil((float)T / dimBlock.y),
                1 };

    err = hipMalloc(&in_d, width * height * sizeof(double));
    CUDA_CHECK(err, "can't hipMalloc!");
    err = hipMalloc(&out_d, width * height * sizeof(double));
    CUDA_CHECK(err, "can't hipMalloc!");

    err = hipMemcpy(
      in_d, in_h, width * height * sizeof(double), hipMemcpyHostToDevice);
    CUDA_CHECK(err, "Can't hipMemcpy!");

    matTransposeKernel<<<dimGrid, dimBlock>>>(in_d, out_d, width, height);

    err = hipMemcpy(
      out_h, out_d, width * height * sizeof(double), hipMemcpyDeviceToHost);
    CUDA_CHECK(err, "Can't hipMemcpy!");

Error:
    hipFree(in_d);
    hipFree(out_d);

    return err;
}
